#include "hip/hip_runtime.h"

#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include "types.h"
#include <hip/hip_vector_types.h>


#define BLOCKDIM_H 16
#define BLOCKDIM_W 16


namespace cg = cooperative_groups;


__global__ void ConvolutionKernel(
	BYTE * DestinationMatrix,
	BYTE * SourceMatrix,
	int ImageWidth,
	int ImageHeight,
	BYTE * KernelMatrix,
	int KernelRadius
)
{
	cg::thread_block cta = cg::this_thread_block();
	extern __shared__ BYTE blockMatrix[];
	// matrice cu width BLOCKDIM_X + 2*kernelRadius heigth BLOCKDIM_Y + 2*kernelRadius

	int blockIndex;
	int sourceIndexX, sourceIndexY, sourceIndex;
	int blockWidth = BLOCKDIM_W + 2 * KernelRadius;
	int blockHeight = BLOCKDIM_H + 2 * KernelRadius;
	int x, y;

	//load pixel
	blockIndex = threadIdx.y * blockHeight + threadIdx.x;
	sourceIndexX = blockIdx.x * BLOCKDIM_W + threadIdx.x - KernelRadius;
	sourceIndexY = blockIdx.y * BLOCKDIM_H + threadIdx.y - KernelRadius;
	sourceIndex = sourceIndexY * ImageWidth + sourceIndexX;
	blockMatrix[blockIndex] = 0;
	blockMatrix[blockIndex] = (sourceIndexX >= 0 && sourceIndex < ImageWidth && sourceIndexY >= 0 && sourceIndexY < ImageHeight)? SourceMatrix[sourceIndex] : 0;

	cg::sync(cta);

	if (threadIdx.x < BLOCKDIM_W + KernelRadius && threadIdx.x >= KernelRadius &&
		threadIdx.y < BLOCKDIM_H + KernelRadius && threadIdx.y >= KernelRadius)
	{
		BYTE accumulator = 0;

		for (int kernelX = -KernelRadius; kernelX <= KernelRadius; kernelX++)
		{

			for (int kernelY = -KernelRadius; kernelY <= KernelRadius; kernelY++)
			{

				accumulator += KernelMatrix[(kernelY + KernelRadius) * (2 * KernelRadius + 1) + (kernelY + KernelRadius)] * 
						blockMatrix[(threadIdx.y + kernelY) * blockWidth + (threadIdx.x + kernelX)];
			}
		}

		DestinationMatrix[sourceIndex] = accumulator;
	}

}

extern "C"
void
ConvolutionGPU(
	BYTE * DestinationMatrix,
	BYTE * SourceMatrix,
	int ImageWidth,
	int ImageHeight,
	BYTE * KernelMatrix,
	int KernelRadius
)
{
	assert(ImageWidth % BLOCKDIM_W == 0);
	assert(ImageHeight % BLOCKDIM_H == 0);

	dim3 blocks(ImageWidth / BLOCKDIM_W, ImageHeight / (BLOCKDIM_H));
	dim3 threads(BLOCKDIM_W + 2*KernelRadius, BLOCKDIM_H + 2*KernelRadius);


	ConvolutionKernel <<<blocks, threads, (BLOCKDIM_W + 2*KernelRadius)*(BLOCKDIM_H + 2*KernelRadius) >>> (
		DestinationMatrix,
		SourceMatrix,
		ImageWidth,
		ImageHeight,
		KernelMatrix,
		KernelRadius
	);
}
