
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "types.h"
#include "fileUtils.h"

hipError_t
convolutionWithCuda(
	PBYTE *	DestinationMatrix,
	BYTE *	SourceMatrix,
	int ImageWidth,
	int ImageHeight,
	BYTE * KernelMatrix,
	int KernelRadius
);

extern "C"
void
ConvolutionGPU(
	BYTE * DestinationMatrix,
	BYTE * SourceMatrix,
	int ImageWidth,
	int ImageHeight,
	BYTE * KernelMatrix,
	int KernelRadius
);

int main()
{
    PBYTE matrix;
	PBYTE kernel;
	unsigned int matrixWidth, matrixHeight;
	unsigned int kernelRadius, kernelLength;
	PBYTE resultMatrix;

	if (!ReadSampleMatrix(&matrix, &matrixWidth, &matrixHeight))
	{
		return 1;
	}

	if (!ReadKernel(&kernel, &kernelLength, NULL))
	{
		return 1;
	}
	
	kernelRadius = (kernelLength - 1) / 2;

	printf("Matrices read\n");

    hipError_t cudaStatus = convolutionWithCuda(&resultMatrix, matrix, matrixWidth, matrixHeight, kernel, kernelRadius);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "convolutionWithCuda failed!");
        return 1;
    }

    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
    return 0;
}

hipError_t
convolutionWithCuda(
	PBYTE *	DestinationMatrix,
	BYTE *	SourceMatrix,
	int ImageWidth,
	int ImageHeight,
	BYTE * KernelMatrix,
	int KernelRadius
)
{
	BYTE *deviceSourceMatrix = NULL;
    BYTE *deviceDestinationMatrix = NULL;
	BYTE *resultMatrix = NULL;	
    BYTE *kernel = NULL;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		fprintf(stderr, "%d\n", cudaStatus);
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	resultMatrix = (PBYTE)malloc(ImageWidth * ImageHeight * sizeof(BYTE));
	
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&deviceSourceMatrix, ImageHeight * ImageWidth * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&deviceDestinationMatrix, ImageHeight * ImageWidth * sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&kernel, (KernelRadius * 2 + 1)* (KernelRadius * 2 + 1)* sizeof(BYTE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(deviceSourceMatrix, SourceMatrix, ImageHeight * ImageWidth * sizeof(BYTE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(kernel, KernelMatrix, (KernelRadius * 2 + 1)* (KernelRadius * 2 + 1) * sizeof(BYTE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
	ConvolutionGPU(
		deviceDestinationMatrix, deviceSourceMatrix, ImageWidth, ImageHeight, kernel, KernelRadius
	);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(resultMatrix, deviceDestinationMatrix, ImageHeight * ImageWidth * sizeof(BYTE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(deviceDestinationMatrix);
    hipFree(deviceSourceMatrix);
    hipFree(kernel);
    
	*DestinationMatrix = resultMatrix;

    return cudaStatus;
}
